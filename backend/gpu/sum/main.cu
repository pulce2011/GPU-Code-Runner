#include <hip/hip_runtime.h>

#include "sum.cuh"

int main() {
    const int n_threads = 1024;
    const int N = 10240;
    const int F = 35;
    float a_h[N], b_h[N], c_h[N], c_h2[N];
    float *a_d, *b_d, *c_d;
    for (int i=0; i<N; i++) {
        a_h[i] = i/35;
        b_h[i] = (i+F)/100;
        c_h2[i] = a_h[i] + b_h[i];
    }
    hipMalloc(&a_d, sizeof(float)*N);
    hipMalloc(&b_d, sizeof(float)*N);
    hipMalloc(&c_d, sizeof(float)*N);
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(float)*N, hipMemcpyHostToDevice);

    sum<<<n_threads, N/n_threads>>>(a_d, b_d, c_d, N);

    hipMemcpy(c_h, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++) {
        if (c_h2[i] != c_h[i]) {
            return 1;
        }
    }


    return 0;
}